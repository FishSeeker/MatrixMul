#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "hip/hip_fp16.h"
#include "fp16_conversion.h"



#ifndef min
#define min(a,b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif
#define BLOCK_SIZE 32

typedef struct _matrixSize      // Optional Command-line multiplier for matrix sizes
{
    unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
} sMatrixSize;

__global__ void shared_kernel(float *C, float *A, float *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
       __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
       __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

__global__ void general_kernel(float *C,const float *A, const float *B, int N,int M,int S)
{
    int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x
                    + blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < M * S)
    {
        int row = threadId / S;
        int column = threadId % S;
        float sum = 0.0;
        C[threadId] = 0;
        for (int i = 0; i < N; i++)
        {
             sum += A[row * N + i] * B[i * S + column];
        }
        C[threadId] = sum;
    }
}

void constantInit(float *data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = 0.2;
    }
}

void constantInit_half( half *data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] =  approx_float_to_half(0.2);
    }
}

void cpu(dim3 &dimsA, dim3 &dimsB){
	//dimsA.x 就是列数，dimsB.y是行数
	int hA = dimsA.x,wA = dimsA.y,wB = dimsB.x;
	float *A,*B,*C;
	A = (float*)malloc(sizeof(float) * dimsA.x * dimsA.y);
	B = (float*)malloc(sizeof(float) * dimsB.x * dimsB.y);
	C = (float*)malloc(sizeof(float) * dimsA.y * dimsB.x);

	constantInit(A,dimsA.x * dimsA.y,1.0);
	constantInit(B,dimsB.x * dimsB.y,1.0);
    for (unsigned int i = 0; i < hA; ++i){
        for (unsigned int j = 0; j < wB; ++j)
        {
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k)
            {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
    }
    for(int i=0;i<10;i++)
    	printf("%f ",C[i]);

}

int shared(dim3 &dimsA, dim3 &dimsB)
{
	printf("\nstart=================shared memory================================\n");
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // Initialize host memory
    const float valB = 1.0f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);

    if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    hipError_t error;

    error = hipMalloc((void **) &d_A, mem_size_A);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_B, mem_size_B);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_C, mem_size_C);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    shared_kernel<<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);


    printf("done\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Execute the kernel
    int nIter = 30;

    for (int j = 0; j < nIter; j++)
    {

        shared_kernel<<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);

    }

    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps


    for(int i=0;i<10;i++)
    	printf("%f ",h_C[i]);

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    printf("\nend=================shared memory================================\n");
    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}

int general(dim3 &dimsA, dim3 &dimsB)
{
	printf("\nstart=================general================================\n");
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // Initialize host memory
    const float valB = 1.0f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);

    if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    hipError_t error;

    error = hipMalloc((void **) &d_A, mem_size_A);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_B, mem_size_B);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_C, mem_size_C);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");


    // Performs warmup operation using matrixMul CUDA kernel
    general_kernel<<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsA.y,dimsB.x);
    printf("done\n");
    hipDeviceSynchronize();



    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Execute the kernel
    int nIter = 30;

    for (int j = 0; j < nIter; j++)
    {

        general_kernel<<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsA.y,dimsB.x);

    }

    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps

    for(int i=0;i<10;i++)
    	printf("%f ",h_C[i]);
    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    printf("\nend=================general================================\n");
    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}

int single_cublas(dim3 &dimsA, dim3 &dimsB)
{
	printf("\nstart=================single cublas================================\n");
	sMatrixSize matrix_size;
	//在这对matrix_size根据dimsA和dimsB赋值

	matrix_size.uiHA = dimsA.y;
	matrix_size.uiWA = dimsA.x;
	matrix_size.uiHB = dimsB.y;
	matrix_size.uiWB = dimsB.x;
	matrix_size.uiHC = dimsA.y;
	matrix_size.uiWC = dimsB.x;

    // allocate host memory for matrices A and B
    unsigned int size_A = matrix_size.uiWA * matrix_size.uiHA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = matrix_size.uiWB * matrix_size.uiHB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);


    constantInit(h_A,size_A,1.0);
    constantInit(h_B,size_B,1.0);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    unsigned int size_C = matrix_size.uiWC * matrix_size.uiHC;
    unsigned int mem_size_C = sizeof(float) * size_C;

    // allocate host memory for the result
    float *h_C      = (float *) malloc(mem_size_C);
    float *h_CUBLAS = (float *) malloc(mem_size_C);

    (hipMalloc((void **) &d_A, mem_size_A));
    (hipMalloc((void **) &d_B, mem_size_B));
    (hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    (hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));
    (hipMalloc((void **) &d_C, mem_size_C));

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(matrix_size.uiWC / threads.x, matrix_size.uiHC / threads.y);

    // create and start timer
    printf("Computing result using CUBLAS...");

    // execute the kernel
    int nIter = 30;

    // CUBLAS version 2.0
    {
        const float alpha = 1.0f;
        const float beta  = 0.0f;
        hipblasHandle_t handle;
        hipEvent_t start, stop;

        (hipblasCreate(&handle));

        //Perform warmup operation with cublas
        (hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));

        // Allocate CUDA events that we'll use for timing
        (hipEventCreate(&start));
        (hipEventCreate(&stop));

        // Record the start event
        (hipEventRecord(start, NULL));

        for (int j = 0; j < nIter; j++)
        {
            //note cublas is column primary!
            //need to transpose the order
            (hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));

        }

        printf("done.\n");

        // Record the stop event
        (hipEventRecord(stop, NULL));

        // Wait for the stop event to complete
        (hipEventSynchronize(stop));

        float msecTotal = 0.0f;
        (hipEventElapsedTime(&msecTotal, start, stop));

        // Compute and print the performance
        float msecPerMatrixMul = msecTotal / nIter;
        double flopsPerMatrixMul = 2.0 * (double)matrix_size.uiHC * (double)matrix_size.uiWC * (double)matrix_size.uiHB;
        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
        printf(
            "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
            gigaFlops,
            msecPerMatrixMul,
            flopsPerMatrixMul);

        // copy result from device to host
        (hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost));

        // Destroy the handle
        (hipblasDestroy(handle));
    }
    for(int i=0;i<10;i++)
    	printf("%f ",h_CUBLAS[i]);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    (hipFree(d_A));
    (hipFree(d_B));
    (hipFree(d_C));
    printf("\nend=================single cublas================================\n");
    return 1;

}

int half_cublas(dim3 &dimsA, dim3 &dimsB)
{
	printf("\nstart=================half cublas================================\n");
	sMatrixSize matrix_size;
	//在这对matrix_size根据dimsA和dimsB赋值

	matrix_size.uiHA = dimsA.y;
	matrix_size.uiWA = dimsA.x;
	matrix_size.uiHB = dimsB.y;
	matrix_size.uiWB = dimsB.x;
	matrix_size.uiHC = dimsA.y;
	matrix_size.uiWC = dimsB.x;

    // allocate host memory for matrices A and B
    unsigned int size_A = matrix_size.uiWA * matrix_size.uiHA;
    unsigned int mem_size_A = sizeof(half) * size_A;
    half *h_A = (half *)malloc(mem_size_A);
    unsigned int size_B = matrix_size.uiWB * matrix_size.uiHB;
    unsigned int mem_size_B = sizeof(half) * size_B;
    half *h_B = (half *)malloc(mem_size_B);


    constantInit_half(h_A,size_A,1.0);
    constantInit_half(h_B,size_B,1.0);

    // allocate device memory
    __half *d_A, *d_B, *d_C;
    unsigned int size_C = matrix_size.uiWC * matrix_size.uiHC;
    unsigned int mem_size_C = sizeof(half) * size_C;

    // allocate host memory for the result
    half *h_C      = (half *) malloc(mem_size_C);
    half *h_CUBLAS = (half *) malloc(mem_size_C);

    (hipMalloc((void **) &d_A, mem_size_A));
    (hipMalloc((void **) &d_B, mem_size_B));
    (hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    (hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));
    (hipMalloc((void **) &d_C, mem_size_C));

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(matrix_size.uiWC / threads.x, matrix_size.uiHC / threads.y);

    // create and start timer
    printf("Computing result using CUBLAS...");

    // execute the kernel
    int nIter = 30;

    // CUBLAS version 2.0
    {
    	const __half alpha = approx_float_to_half(1.0);
    	const __half beta = approx_float_to_half(0.0);


        hipblasHandle_t handle;
        hipEvent_t start, stop;

        (hipblasCreate(&handle));

        //Perform warmup operation with cublas
        (hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));

        // Allocate CUDA events that we'll use for timing
        (hipEventCreate(&start));
        (hipEventCreate(&stop));

        // Record the start event
        (hipEventRecord(start, NULL));

        for (int j = 0; j < nIter; j++)
        {
            //note cublas is column primary!
            //need to transpose the order
            (hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));
        }

        printf("done.\n");

        // Record the stop event
        (hipEventRecord(stop, NULL));

        // Wait for the stop event to complete
        (hipEventSynchronize(stop));

        float msecTotal = 0.0f;
        (hipEventElapsedTime(&msecTotal, start, stop));

        // Compute and print the performance
        float msecPerMatrixMul = msecTotal / nIter;
        double flopsPerMatrixMul = 2.0 * (double)matrix_size.uiHC * (double)matrix_size.uiWC * (double)matrix_size.uiHB;
        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
        printf(
            "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
            gigaFlops,
            msecPerMatrixMul,
            flopsPerMatrixMul);

        // copy result from device to host
        (hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost));

        // Destroy the handle
        (hipblasDestroy(handle));
    }
    for(int i=0;i<10;i++)
    	printf("%f ",half_to_float(h_CUBLAS[i]));

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    (hipFree(d_A));
    (hipFree(d_B));
    (hipFree(d_C));
    printf("\nend=================half cublas================================\n");
    return 1;
}


int main(int argc, char **argv)
{
	int multiple = 64;
    dim3 dimsA(multiple*BLOCK_SIZE, multiple*BLOCK_SIZE, 1);
    dim3 dimsB(multiple*BLOCK_SIZE, multiple*BLOCK_SIZE, 1);


    if (dimsA.x != dimsB.y)
    {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);


    // cpu(dimsA, dimsB);
    shared(dimsA, dimsB);
    general(dimsA, dimsB);
    single_cublas(dimsA, dimsB);
    half_cublas(dimsA, dimsB);

    return 0;
}
